#include <stddef.h>  // NULL, size_t
#include <math.h> // expf
#include <stdio.h> // printf
#include <time.h> // time
#include <sys/time.h> // gettimeofday
#include <assert.h>



#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
//#include "cutil.h" // CUDA_SAFE_CALL, CUT_CHECK_ERROR


#include <iostream>
#include <fstream>


using namespace std;




#define NUM_BLOCKS 1000
#define NUM_THREADS 1000  // threads per block
#define N 1000
#define DT 0.001


#define SEED (time(NULL)) // random seed





/***
 * Device functions
 ***/


__global__ void initialize(float *dev_W0){

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int tt = bid*NUM_BLOCKS+tid;

  dev_W0[tt] = 0.0;
}


__global__ void update_W(float *dev_W0, float *dev_W){

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int tt = bid*NUM_BLOCKS+tid;

  dev_W0[tt] = dev_W0[tt]+sqrt(DT)*dev_W[tt];

}


/*
main function
*/

int main(void)
{
  
  int i,j;

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);


  // this is for timing
  clock_t starttime=clock();

  // allocate memory
  float **W;
	W = (float**)malloc(N*sizeof(float*));
  if (W==NULL)
    {
      printf("Can't memalloc W\n");
      return 0;
    }
  
  for (i=0; i<N; i++)
    {
      W[i]=(float*)malloc((NUM_BLOCKS*NUM_THREADS)*sizeof(float));
      if (W[i]==NULL)
        {
          printf("Can't memalloc W[%d]\n",i);
          return 0;
        }
    }



  float *dev_W;
  hipMalloc( (void**)&dev_W, NUM_BLOCKS*NUM_THREADS*sizeof(float) );
  
  float *dev_W0;
  hipMalloc( (void**)&dev_W0, NUM_BLOCKS*NUM_THREADS*sizeof(float) );

  initialize<<<NUM_BLOCKS,NUM_THREADS>>>(dev_W0);
  hipMemcpy(W[0], dev_W0, NUM_BLOCKS*NUM_THREADS*sizeof(float), hipMemcpyDeviceToHost);
  for(i=1;i<N;i++){
    hiprandGenerateNormal(gen, dev_W, NUM_BLOCKS*NUM_THREADS, 0.0,1.0);
    update_W<<<NUM_BLOCKS,NUM_THREADS>>>(dev_W0, dev_W);
    hipMemcpy(W[i], dev_W0, NUM_BLOCKS*NUM_THREADS*sizeof(float), hipMemcpyDeviceToHost);
  }



  hipFree(dev_W0);
  hipFree(dev_W);

  printf("Time elapsed: %f \n",  ((double)clock() - starttime)/CLOCKS_PER_SEC);
    


  printf("Saving a few paths to a file ... \n");
  int myidx[10] = {12, 3234, 534534, 534, 45345, 3434, 999999, 13135, 38, 89343};
  FILE *file1;
  FILE *file2;

  file1 = fopen("path_matrix.txt","w");
  for(j=0;j<N;j++){
    for(i=0;i<10;i++){
        fprintf(file1,"%.6f ", W[j][ myidx[i] ]);
    }
    fprintf(file1, "\n");
  }
  fclose(file1);


  // check to see if W(0.5) and W(1) have the correct distributions
  file2 = fopen("for_hist.txt","w");
  for(j=0;j<1000000;j++){
    fprintf(file2, "%.6f, ", W[499][j]);
    fprintf(file2, "%.6f\n", W[999][j]);
  }
  fclose(file2);


  free(W);

  return 1;
}